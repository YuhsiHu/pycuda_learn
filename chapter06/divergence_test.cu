#include <hip/hip_runtime.h>
#include <stdio.h>

/**
编写一个程序来了解分支处理机制
nvcc divergence_test.cu -o divergence_test
*/
__global__ void divergence_test_ker()
{
	if( threadIdx.x % 2 == 0)
		printf("threadIdx.x %d : This is an even thread.\n", threadIdx.x);
	else
		printf("threadIdx.x %d : This is an odd thread.\n", threadIdx.x);
}

__host__ int main()
{
	hipSetDevice(0);
	divergence_test_ker<<<1, 32>>>();
	hipDeviceSynchronize();
	hipDeviceReset();
}