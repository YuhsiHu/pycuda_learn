#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define _EPSILON 0.001
#define _ABS(x)	( x > 0.0f ? x : -x )

/**
compile: nvcc matrix_ker.cu -o matrix_ker
run: ./matrix_ker
*/

// 加上__host__表示该函数运行在CPU上
__host__ int allclose(float *A, float *B, int len)
{

	int returnval = 0;
	
	for (int i = 0; i < len; i++)
	{
		if ( _ABS(A[i] - B[i]) > _EPSILON )
		{
			returnval = -1;
			break;
		}
	}
	
	return(returnval);
}

// row-column dot-product for matrix multiplication
__device__ float rowcol_dot(float *matrix_a, float *matrix_b, int row, int col, int N)
{
	float val = 0;
	
	for (int k=0; k < N; k++)
	{
        val += matrix_a[ row*N + k ] * matrix_b[ col + k*N];
	}
	
	return(val);
}

// matrix multiplication kernel that is parallelized over row/column tuples.
__global__ void matrix_mult_ker(float * matrix_a, float * matrix_b, float * output_matrix, int N)
{

    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

	output_matrix[col + row*N] = rowcol_dot(matrix_a, matrix_b, row, col, N);
}

__host__ int main()
{

	// Initialize to use first GPU.
	hipSetDevice(0);

	// this indicates the width/height of the matrices
	int N = 4;
	
	// this will indicate how many bytes to allocate to store a test or output matrix
	int num_bytes = sizeof(float)*N*N;
	
	// input test matrix A
	float h_A[] = {	1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0, \
					1.0,  2.0,  3.0,  4.0 };
					
	// input test matrix B
	float h_B[] = {	14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0, \
					14.0,  13.0,  12.0,  11.0 };
	
	// expected output of A times B
	float h_AxB[] = { 140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0, \
					140.0,  130.0,  120.0,  110.0 };
					
					
	// these pointers will be used for the GPU.
	// (notice how we use normal float pointers)
	float * d_A;
	float * d_B;
	float * d_output;
	
	// allocate memory for the test matrices on the GPU
	hipMalloc((float **) &d_A, num_bytes);
	hipMalloc((float **) &d_B, num_bytes);
	
	// copy the test matrices to the GPU
	hipMemcpy(d_A, h_A, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, num_bytes, hipMemcpyHostToDevice);
	
	// allocate memory for output on GPU
	hipMalloc((float **) &d_output, num_bytes);
	
	// this will store the output from the GPU
	float * h_output;
	h_output = (float *) malloc(num_bytes);

	// setup our block and grid launch parameters with the dim3 class.
	dim3 block(2,2,1);
	dim3 grid(2,2,1);
	
	// launch our kernel
	matrix_mult_ker <<< grid, block >>> (d_A, d_B, d_output, N);
	
	// synchronize on the host, to ensure our kernel has finished executing.
	hipDeviceSynchronize();
	
	// copy output from device to host.
	hipMemcpy(h_output, d_output, num_bytes, hipMemcpyDeviceToHost);

	// synchronize again.
	hipDeviceSynchronize();
	
	// free arrays on device.
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_output);
	
	// reset the GPU.
	hipDeviceReset();
	
	// Check to see if we got the expected output.
	// in both cases, remember to de-allocate h_output before returning.
	if (allclose(h_AxB, h_output, N*N) < 0)
	{
		printf("Error!  Output of kernel does not match expected output.\n");
		free(h_output);
		return(-1);
	}
	else
	{
		printf("Success!  Output of kernel matches expected output.\n");
		free(h_output);
		return(0);
	}

}